#include "hip/hip_runtime.h"
#include "dawn.hpp"
using namespace std;

__global__ void vecMatOpe(int rows, int *d_A_entry, int *d_A, bool *input, bool *output, int *result, int source, int dim, int *entry);
void runApspGpu(DAWN::Graph &matrix, std::string &output_path);
float sssp_gpu(DAWN::Graph &matrix, int source, hipStream_t streams, int *d_A_entry, int *d_A, int *&result);

void runApspGpu(DAWN::Graph &matrix, std::string &output_path)
{
    std::ofstream outfile(output_path);
    if (!outfile.is_open())
    {
        std::cerr << "Error opening file " << output_path << std::endl;
        return;
    }
    float elapsed_time = 0.0;
    int proEntry = 0;

    omp_set_dynamic(true);
    DAWN dawn;

    std::cerr << "分配显存" << std::endl;

    // Copy data to device
    int *d_A_entry, *d_A;
    hipMalloc((void **)&d_A_entry, sizeof(int) * matrix.rows);
    hipMalloc((void **)&d_A, sizeof(int) * matrix.nnz);

    cout << matrix.nnz << endl;
    int *h_A = new int[matrix.nnz];
    int tmp = 0;
    for (int i = 0; i < matrix.rows; i++)
    {
        if (matrix.A_entry[i] == 0)
        {
            matrix.A_entry[i] = matrix.A_entry[i - 1];
            continue;
        }
        for (int j = 0; j < matrix.A_entry[i]; j++)
        {
            h_A[tmp] = matrix.A[i][j];
            // cout << h_A[tmp] << endl;
            tmp++;
        }
        matrix.A_entry[i] = tmp;
        // cout << matrix.A_entry[i] << endl;
    }

    std::cerr << "复制矩阵" << std::endl;

    hipMemcpy(d_A_entry, matrix.A_entry, sizeof(int) * matrix.rows, hipMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, sizeof(int) * matrix.nnz, hipMemcpyHostToDevice);

    // for (int i = 0; i < matrix.nnz; i++)
    // {
    //     printf("A[%d] = %d\n", i, h_A[i]); // 打印元素值
    // }

    // Create streams
    hipStream_t streams[matrix.stream];
    for (int i = 0; i < matrix.stream; i++)
    {
        hipStreamCreate(&streams[i]);
    }

    std::cout << ">>>>>>>>>>>>>>>>>>>>>>>>>>> APSP start <<<<<<<<<<<<<<<<<<<<<<<<<<<" << std::endl;

    for (int i = 0; i < matrix.rows; i++)
    {

        int source = i;
        if (matrix.B_entry[i] == 0)
        {
            proEntry++;
            dawn.infoprint(proEntry, matrix.rows, matrix.interval, matrix.thread, elapsed_time);
            continue;
        }
        int hip_stream = source % matrix.stream;
        float time_tmp = 0.0f;
        int *result = new int[matrix.rows];
        // if (i == 5)
        time_tmp = sssp_gpu(matrix, source, streams[hip_stream], d_A_entry, d_A, result);
        elapsed_time += time_tmp;
        proEntry++;
        dawn.infoprint(proEntry, matrix.rows, matrix.interval, matrix.thread, elapsed_time);
        // for (int j = 0; j < matrix.rows; j++)
        // {
        //     if (i != j)
        //         outfile << i << " " << j << " " << result[j] << endl;
        // }
        delete[] result;
        result = nullptr;
    }
    std::cout << ">>>>>>>>>>>>>>>>>>>>>>>>>>> APSP end <<<<<<<<<<<<<<<<<<<<<<<<<<<" << std::endl;
    // Output elapsed time and free remaining resources
    std::cout << " Elapsed time: " << elapsed_time / (matrix.thread * 1000) << std::endl;

    outfile.close();

    // Synchronize streams
    for (int i = 0; i < matrix.stream; i++)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }

    // Free memory on device
    hipFree(d_A_entry);
    hipFree(d_A);
}

float sssp_gpu(DAWN::Graph &matrix, int source, hipStream_t streams, int *d_A_entry, int *d_A, int *&result)
{
    int dim = 1;
    int entry = matrix.B_entry[source];
    int entry_last = entry;
    bool *output = new bool[matrix.rows];
    bool *input = new bool[matrix.rows];
    omp_set_dynamic(true);
#pragma omp parallel for
    for (int j = 0; j < matrix.rows; j++)
    {
        input[j] = false;
        output[j] = false;
        result[j] = 0;
    }
#pragma omp parallel for
    for (int i = 0; i < matrix.B_entry[source]; i++)
    {
        input[matrix.B[source][i]] = true;
        output[matrix.B[source][i]] = true;
        result[matrix.B[source][i]] = 1;
    }

    bool *d_input, *d_output;
    int *d_result;
    int *d_entry;
    // int *d_dim, *d_source;
    hipMalloc((void **)&d_input, sizeof(bool) * matrix.cols);
    hipMalloc((void **)&d_output, sizeof(bool) * matrix.rows);
    hipMalloc((void **)&d_result, sizeof(int) * matrix.rows);
    hipMalloc(&d_entry, sizeof(int));
    // hipMalloc(&d_dim, sizeof(int));
    // hipMalloc(&d_source, sizeof(int));

    hipMemcpyAsync(d_input, input, sizeof(bool) * matrix.rows, hipMemcpyHostToDevice, streams);
    hipMemcpyAsync(d_output, output, sizeof(bool) * matrix.rows, hipMemcpyHostToDevice, streams);
    hipMemcpyAsync(d_entry, &entry, sizeof(int), hipMemcpyHostToDevice, streams);
    hipMemcpyAsync(d_result, result, sizeof(int) * matrix.rows, hipMemcpyHostToDevice, streams);
    // Launch kernel
    int block_size = matrix.block_size;
    int num_blocks = (matrix.cols + block_size - 1) / block_size;

    auto start = std::chrono::high_resolution_clock::now();
    while (dim < matrix.dim)
    {
        dim++;
        // cout << endl;
        // cout << "计算前：" << endl;
        // cout << "input:" << endl;
        // for (int j = 0; j < matrix.rows; j++)
        // {
        //     cout << input[j] << " ";
        // }
        // cout << "output:" << endl;
        // for (int j = 0; j < matrix.rows; j++)
        // {
        //     cout << output[j] << " ";
        // }
        // cout << endl;

        vecMatOpe<<<num_blocks, block_size, 0, streams>>>(matrix.rows, d_A_entry, d_A, d_input, d_output, d_result, source, dim, d_entry);
        // vecMatOpe(int rows, int *d_A_entry, int *d_A, bool *input, bool *output, int *result, int source, int dim, int *entry);

        // cout << "计算后：" << endl;
        // cout << "input:" << endl;
        // for (int j = 0; j < matrix.rows; j++)
        // {
        //     cout << input[j] << " ";
        // }
        // cout << "output:" << endl;
        // for (int j = 0; j < matrix.rows; j++)
        // {
        //     cout << output[j] << " ";
        // }
        // cout << endl;
        // #pragma omp parallel for
        //         for (int j = 0; j < matrix.rows; j++)
        //         {
        //             if ((result[j] == 0) && (output[j] == true) && (source != j))
        //             {
        //                 result[j] = dim;
        // #pragma omp critical
        //                 {
        //                     entry++;
        //                 }
        //             }
        //             input[j] = output[j];
        //             output[j] = false;
        //         }
        hipMemcpyAsync(&entry, d_entry, sizeof(int), hipMemcpyDeviceToHost, streams);

        if ((entry > entry_last) && (entry < matrix.rows))
        {
            entry_last = entry;
            if (entry_last >= matrix.rows - 1) // entry = matrix.rows - 1意味着向量填满，无下一轮
                break;
        }
        else // 如果没有新的最短路径产生，则退出循环
        {
            break;
        }
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    hipMemcpyAsync(result, d_result, sizeof(int) * matrix.rows, hipMemcpyDeviceToHost, streams);
    matrix.entry += entry_last;

    delete[] output;
    output = nullptr;
    delete[] input;
    input = nullptr;

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_result);
    hipFree(d_entry);

    return elapsed.count();
}

__global__ void vecMatOpe(int rows, int *d_A_entry, int *d_A, bool *input, bool *output, int *result, int source, int dim, int *entry)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // if (j < rows)
    // {
    //     printf("input[%d] = %d\n", j, input[j]);
    // }
    // if (j < rows)
    // {
    //     printf("output[%d] = %d\n", j, output[j]);
    // }

    if (j < rows && (d_A_entry[j] != d_A_entry[j - 1]))
    {

        int start = (j == 0 ? 0 : d_A_entry[j - 1]); // 当前行的起始位置
        int end = d_A_entry[j];                      // 当前行的结束位置
        for (int k = start; k < end; k++)
        { // 索引矩阵A的当前行
          // printf("A[%d] = %d\n", k, d_A[k]); // 打印元素值
            if (input[d_A[k]] == true)
            {
                // printf("A[%d] = %d\n", k, d_A[k]);
                // printf("input[%d] = %d\n", d_A[k], input[d_A[k]]);
                output[j] = true;
                break;
            }
        }
        // printf("A_entry[%d] = %d\n", j, d_A_entry[j]); // 打印元素值
    }
    if (j < rows)
    {
        if ((result[j] == 0) && (output[j] == true) && (source != j))
        {
            result[j] = dim;
            atomicAdd(entry, 1); // use atomic add to ensure entry is incremented safely
        }
        input[j] = output[j];
        output[j] = false;
    }
    // if (j < rows)
    // {
    //     printf("input[%d] = %d\n", j, input[j]);
    // }
    // if (j < rows)
    // {
    //     printf("output[%d] = %d\n", j, output[j]);
    // }
}

int main(int argc, char *argv[])
{
    string input_path = argv[1];
    string output_path = argv[2];
    int stream = atoi(argv[3]);
    int block_size = atoi(argv[4]);

    std::ifstream file(input_path);
    if (!file.is_open())
    {
        std::cerr << "Error opening file " << input_path << std::endl;
        return;
    }

    DAWN dawn;
    DAWN::Graph matrix;
    matrix.thread = 1; // 运行SSSP的线程,GPU版本默認版本
    matrix.interval = 100;
    matrix.stream = stream;         // 32
    matrix.block_size = block_size; // 4
    dawn.createGraph(input_path, matrix);
    runApspGpu(matrix, output_path);

    return 0;
}